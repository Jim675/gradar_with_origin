#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>

#include "constant.h"

using std::cout;
using std::endl;

//constexpr double PI = 3.1415926535897932384626433832795;
//
//// ����뾶
//constexpr double RE = 6371393.0;
//
//// ��Ч����뾶
//constexpr double RM = RE * 4.0 / 3.0;
//
//// �״ﴫ��Բ��·���İ뾶Ϊ����뾶4��
//constexpr double RN = RE * 4.0;
//
//// 2����Ч����뾶
////constexpr double RM_2 = RM * 2;
//
//// ���(equator)�ܳ�(circumference)  Equatorial Circumference
//constexpr double EC = 20037508.3427892430765884088807;
//
//

// CUDA kernel����, ������GPU֮��
__global__ void kernel_calc(double* d_x, double* d_y, double* d_az, double* d_r,
                            const double slon, const double slat, const double el, const double cv,
                            const unsigned int height, const unsigned int width)
{
    const unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= height * width) {
        return;
    }
    const unsigned int row = index / width;
    const unsigned int col = index % width;
    // ī����ֱ��ת��γ�Ȼ���
    const double dlon = d_x[col] / EC * PI - slon;
    const double lat = atan(exp(d_y[row] * PI / EC)) * 2.0 - PI / 2.0;
    /*printf("cuda: lon: %lf, lat: %lf\n", lon, lat);*/
    double sin_slat;
    double cos_slat;
    sincos(slat, &sin_slat, &cos_slat);// cuda�����������ù�
    double sin_lat;
    double cos_lat;
    sincos(lat, &sin_lat, &cos_lat);
    // ����γ��֮���Բ�Ľ�
    const double a = acos(sin_slat * sin_lat +
                          cos_slat * cos_lat * cos(dlon));
    const double sin_a = sin(a);
    // ����·������
    const double r = fabs(RN * (a + el + asin(cv * sin_a - sin(a + el))));
    d_r[index] = r;
    if (r >= 250.0) {
        // ��֪��γ����λ��, ����azʵ������sin(az)��Ϊ��ʡ�Ĵ���
        double az = cos_lat * sin(dlon) / sin_a;
        if (az >= -1.0) {
            if (az <= 1.0) {
                az = asin(az);
            } else {
                az = PI / 2;
            }
        } else {
            az = -PI / 2;
        }
        const double dlat = lat - slat;
        // ������λ��
        if (dlon >= 0) {
            if (dlat < 0) az = PI - az;
        } else {
            if (dlat >= 0) az += (2 * PI);
            else az = PI - az;
        }
        d_az[index] = az * 180.0 / PI;
    } else {
        d_az[index] = -1.0;
    }
    //printf("cuda: az: %lf, r: %lf\n", az, r);
}

// CUDA kernel����, ������GPU֮��
__global__ void kernel_calc2(double* d_x, double* d_y, double* d_az, double* d_r,
                             const double slon, const double slat, const double el, const double cv,
                             const unsigned int height, const unsigned int width)
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= height) {
        return;
    }
    // ī����ֱ��ת��γ�Ȼ���
    const double lat = atan(exp(d_y[i] * PI / EC)) * 2.0 - PI / 2.0;
    double sin_slat;
    double cos_slat;
    sincos(slat, &sin_slat, &cos_slat);// cuda�����������ù�
    double sin_lat;
    double cos_lat;
    sincos(lat, &sin_lat, &cos_lat);
    const double dlat = lat - slat;
    for (unsigned int j = 0; j < width; j++) {
        const double dlon = d_x[j] / EC * PI - slon;
        /*printf("cuda: lon: %lf, lat: %lf\n", lon, lat);*/
        // ����γ��֮���Բ�Ľ�
        const double a = acos(sin_slat * sin_lat +
                              cos_slat * cos_lat * cos(dlon));
        const double sin_a = sin(a);
        // ����·������
        const double r = fabs(RN * (a + el + asin(cv * sin_a - sin(a + el))));
        d_r[i * width + j] = r;

        if (r >= 250.0) {
            // ��֪��γ����λ��, ����azʵ������sin(az)��Ϊ��ʡ�Ĵ���
            double az = cos_lat * sin(dlon) / sin_a;
            if (az >= -1.0) {
                if (az <= 1.0) {
                    az = asin(az);
                } else {
                    az = PI / 2;
                }
            } else {
                az = -PI / 2;
            }
            // ������λ��
            if (dlon >= 0) {
                if (dlat < 0) az = PI - az;
            } else {
                if (dlat >= 0) az += (2 * PI);
                else az = PI - az;
            }
            d_az[i * width + j] = az * 180.0 / PI;
        } else {
            d_az[i * width + j] = -1.0;
        }
        //printf("cuda2: az: %lf, r: %lf\n", az, r);
    }
}

static void printDTime(std::chrono::steady_clock::time_point start, std::chrono::steady_clock::time_point end, const char* msg)
{
    auto dtime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    cout << msg << ": " << dtime.count() << "ms" << endl;
}

// ʹ��CUDA���㷽λ�Ǻ;���
extern "C" int calc_az_r(double* x, double* y,
                         double* az, double* r,
                         const double slon, const double slat, const double el, const double elev_add_RE_div_RN,
                         const size_t width, const size_t height)
{

    auto t0 = std::chrono::steady_clock::now();
    // �����豸(GPU)�ϵ�����
    double* d_x = nullptr;
    double* d_y = nullptr;
    double* d_az = nullptr;
    double* d_r = nullptr;

    const size_t n = width * height;
    const size_t wsize = width * sizeof(double);
    const size_t hsize = height * sizeof(double);
    const size_t tsize = n * sizeof(double);

    // �����Դ�
    hipError_t error = hipMalloc(&d_x, wsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_y, hsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_az, tsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_r, tsize);
    if (error != hipSuccess) {
        return error;
    }
    // �����ݴ��ڴ濽�����Դ�
    error = hipMemcpy(d_x, x, wsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMemcpy(d_y, y, hsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }
    // Thread -> Block -> Grid
    //dim3 blocksPerGrid(1);
    //unsigned N = 1024;
    //dim3  threadsPerBlock(N);
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    auto t1 = std::chrono::steady_clock::now();
    printDTime(t0, t1, "CUDA t0 -> t1 spend time");
    kernel_calc << < blocksPerGrid, threadsPerBlock >> > (d_x, d_y, d_az, d_r,
                                                          slon, slat, el, elev_add_RE_div_RN,
                                                          (unsigned int)width, (unsigned int)height);
    hipDeviceSynchronize();
    auto t2 = std::chrono::steady_clock::now();
    printDTime(t1, t2, "CUDA t1 -> t2 spend time");
    error = hipGetLastError();
    if (error != hipSuccess) {
        return error;
    }
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    error = hipMemcpy(az, d_az, tsize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "���ݿ������ڴ�ʧ��" << endl;
        return error;
    }
    error = hipMemcpy(r, d_r, tsize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "���ݿ������ڴ�ʧ��" << endl;
        return error;
    }
    // �ͷ��Դ�
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_az);
    hipFree(d_r);
    // �ͷ��ڴ�
    auto t3 = std::chrono::steady_clock::now();
    printDTime(t2, t3, "CUDA t2 -> t3 spend time");
    printDTime(t0, t3, "CUDA spend time");
    return 0;
}

// ʹ��CUDA���㷽λ�Ǻ;���
extern "C" int calc_az_r2(double* x, double* y,
                          double* az, double* r,
                          const double slon, const double slat, const double el, const double elev_add_RE_div_RN,
                          const size_t width, const size_t height)
{

    auto t0 = std::chrono::steady_clock::now();
    // �����豸(GPU)�ϵ�����
    double* d_x = nullptr;
    double* d_y = nullptr;
    double* d_az = nullptr;
    double* d_r = nullptr;

    const size_t n = width * height;
    const size_t wsize = width * sizeof(double);
    const size_t hsize = height * sizeof(double);
    const size_t tsize = n * sizeof(double);

    // �����Դ�
    hipError_t error = hipMalloc(&d_x, wsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_y, hsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_az, tsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_r, tsize);
    if (error != hipSuccess) {
        return error;
    }
    // �����ݴ��ڴ濽�����Դ�
    error = hipMemcpy(d_x, x, wsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMemcpy(d_y, y, hsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }

    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (height + threadsPerBlock - 1) / threadsPerBlock;
    auto t1 = std::chrono::steady_clock::now();
    printDTime(t0, t1, "CUDA2 t0 -> t1 spend time");
    kernel_calc2 << < blocksPerGrid, threadsPerBlock >> > (d_x, d_y, d_az, d_r,
                                                           slon, slat, el, elev_add_RE_div_RN,
                                                           (unsigned int)width, (unsigned int)height);
    hipDeviceSynchronize();
    auto t2 = std::chrono::steady_clock::now();
    printDTime(t1, t2, "CUDA2 t1 -> t2 spend time");
    error = hipGetLastError();
    if (error != hipSuccess) {
        return error;
    }
    error = hipMemcpy(az, d_az, tsize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "���ݿ������ڴ�ʧ��" << endl;
        return error;
    }
    error = hipMemcpy(r, d_r, tsize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        cout << "���ݿ������ڴ�ʧ��" << endl;
        return error;
    }
    // �ͷ��Դ�
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_az);
    hipFree(d_r);
    auto t3 = std::chrono::steady_clock::now();
    printDTime(t2, t3, "CUDA2 t2 -> t3 spend time");
    printDTime(t0, t3, "CUDA2 spend time");
    return 0;
}

// ʹ��CUDA���㷽λ�Ǻ;���
extern "C" int calc_az_r3(double* x, double* y,
                          double** az, double** r,
                          const double slon, const double slat, const double el, const double elev_add_RE_div_RN,
                          const size_t width, const size_t height)
{

    auto t0 = std::chrono::steady_clock::now();
    // �����豸(GPU)�ϵ�����
    double* d_x = nullptr;
    double* d_y = nullptr;
    //double* d_az = nullptr;
    //double* d_r = nullptr;

    const size_t n = width * height;
    const size_t wsize = width * sizeof(double);
    const size_t hsize = height * sizeof(double);
    const size_t tsize = n * sizeof(double);

    // �����Դ�
    hipError_t error = hipMalloc(&d_x, wsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMalloc(&d_y, hsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMallocManaged(az, tsize);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMallocManaged(r, tsize);
    if (error != hipSuccess) {
        return error;
    }
    // �����ݴ��ڴ濽�����Դ�
    error = hipMemcpy(d_x, x, wsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }
    error = hipMemcpy(d_y, y, hsize, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        return error;
    }
    auto t1 = std::chrono::steady_clock::now();
    // Invoke kernel
    unsigned int threadsPerBlock = 512;
    unsigned int blocksPerGrid = (height + threadsPerBlock - 1) / threadsPerBlock;
    kernel_calc2 << < blocksPerGrid, threadsPerBlock >> > (d_x, d_y, *az, *r,
                                                           slon, slat, el, elev_add_RE_div_RN,
                                                           (unsigned int)width, (unsigned int)height);
    hipDeviceSynchronize();
    auto t2 = std::chrono::steady_clock::now();
    printDTime(t1, t2, "CUDA3 calc kernel spend time");
    error = hipGetLastError();
    if (error != hipSuccess) {
        return error;
    }
    // �ͷ��Դ�
    hipFree(d_x);
    hipFree(d_y);
    auto t3 = std::chrono::steady_clock::now();
    printDTime(t0, t3, "CUDA3 spend time");
    return 0;
}